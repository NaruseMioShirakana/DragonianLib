﻿#include "hip/hip_runtime.h"
#include "kernel.h"

namespace DragonianLib
{
	namespace CudaProvider
	{
		void* cudaAllocate(size_t size)
		{
			void* block = nullptr;
			if (auto err = ::hipMalloc(&block, size))
				fprintf(stderr, "%s\n", hipGetErrorString(err));
			return block;
		}

		int hipFree(void* block)
		{
			return ::hipFree(block);
		}

		int host2Device(void* dst, const void* src, size_t size, stream_t stream)
		{
			return hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, (hipStream_t)stream);
		}

		int device2Host(void* dst, const void* src, size_t size, stream_t stream)
		{
			return hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, (hipStream_t)stream);
		}

		int device2Device(void* dst, const void* src, size_t size, stream_t stream)
		{
			return hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToDevice, (hipStream_t)stream);
		}

		stream_t createCudaStream()
		{
			hipStream_t Ret;
			if (auto err = hipStreamCreate(&Ret))
				fprintf(stderr, "%s\n", hipGetErrorString(err));
			return stream_t(Ret);
		}

		int destoryCudaStream(stream_t stream)
		{
			return hipStreamDestroy((hipStream_t)stream);
		}

		int asyncCudaStream(stream_t stream)
		{
			return hipStreamSynchronize((hipStream_t)stream);
		}

		const char* getCudaError(int errorId)
		{
			return hipGetErrorString(static_cast<hipError_t>(errorId));
		}

		int getLastError()
		{
			return hipGetLastError();
		}
	}
}
