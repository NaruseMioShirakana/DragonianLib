#include "hip/hip_runtime.h"
﻿#include <string>
#include <>

#include "fcpe.h"
#include "hip/hip_runtime.h"

namespace DragonianLib
{
	namespace CudaModules
	{
		namespace FCPE
		{
            ConformerConvModule::ConformerConvModule(
                Module* parent, const std::string& name,
                unsigned dimModel, unsigned expandFactor, unsigned kernelSize
            ) : Module(parent, name)
            {
                auto inner_dim = dimModel * expandFactor;

                net_0 = std::make_shared<LayerNorm1D>(
                    this,
					"net.0",
                    dimModel
                );
                net_2 = std::make_shared<Conv1D>(
                    this,
                    "net.2",
                    dimModel,
                    inner_dim * 2,
                    1
                );
                net_4_conv = std::make_shared<Conv1D>(
                    this,
                    "net.4.conv",
                    inner_dim,
                    inner_dim,
                    kernelSize,
                    1,
                    kernelSize / 2,
					1,
                    inner_dim
                );
                net_6 = std::make_shared<Conv1D>(
                    this,
                    "net.6",
                    inner_dim,
                    dimModel,
                    1
                );
            }

            layerStatus_t ConformerConvModule::Forward(
                Tensor<float>& output,
                Tensor<float>& mean,
                Tensor<float>& var,
                Tensor<float>& cache,
                Tensor<float>& col
            ) const
            {
                if (auto Ret = net_0->Forward(output, mean, var)) return Ret;

                if (auto Ret = net_1.Forward(output, cache)) return Ret;

                if (auto Ret = net_2->Forward(cache, output, col)) return Ret;

                if (auto Ret = net_3.Forward(output, cache)) return Ret;

                if (auto Ret = net_4_conv->Forward(cache, output, col)) return Ret;

                if (auto Ret = net_5.Forward(output)) return Ret;

                if (auto Ret = net_6->Forward(output, cache, col)) return Ret;

                return net_7.Forward(cache, output);
            }

            CFNEncoderLayer::CFNEncoderLayer(
                Module* parent, const std::string& name,
                unsigned dimModel, unsigned numHeads,
                bool useNorm, bool convOnly
            ) : Module(parent, name)
            {
                if (!convOnly)
                    throw std::overflow_error("not impl yet!");

                conformer = std::make_shared<ConformerConvModule>(
                    this,
                    "conformer",
                    dimModel
                );
                norm = std::make_shared<LayerNorm1D>(
                    this,
                    "norm",
                    dimModel
                );
            }

            layerStatus_t CFNEncoderLayer::Forward(
                Tensor<float>& output,
                Tensor<float>& mean,
                Tensor<float>& var,
                Tensor<float>& res,
                Tensor<float>& cache,
                Tensor<float>& col
            ) const
            {
                res.Copy(output);

                if (auto Ret = conformer->Forward(
                    output, mean, var, cache, col
                )) return Ret;

                return AddTensor(output, res);
            }

            ConformerNaiveEncoder::ConformerNaiveEncoder(
                Module* parent, const std::string& name,
                unsigned numLayers, unsigned numHeads, unsigned dimModel,
                bool useNorm, bool convOnly
            ) : Module(parent, name)
            {
                if (!convOnly)
                    throw std::overflow_error("not impl yet!");

                for (unsigned i = 0; i < numLayers; ++i)
                    encoder_layers.emplace_back(
                        std::make_shared<CFNEncoderLayer>(
                            this,
                            "encoder_layers." + std::to_string(i),
                            dimModel,
                            numHeads,
                            useNorm,
                            convOnly
                        )
                    );
            }

            layerStatus_t ConformerNaiveEncoder::Forward(
                Tensor<float>& output,
                Tensor<float>& mean,
                Tensor<float>& var,
                Tensor<float>& res,
                Tensor<float>& cache,
                Tensor<float>& col
            ) const
            {
                for (const auto& layer : encoder_layers)
                    if (auto Ret = layer->Forward(
                        output,
                        mean,
                        var,
                        res,
                        cache,
                        col
                    )) return Ret;
                return LAYER_STATUS_SUCCESS;
            }

			Model::Model(
                unsigned inputChannels, unsigned outputDims, unsigned hiddenDims,
                unsigned numLayers, unsigned numHeads,
                float f0Max, float f0Min,
                bool useFaNorm, bool convOnly,
                bool useHarmonicEmb
            ) : Module(nullptr, "")
            {
                if (!convOnly)
                    throw std::overflow_error("not impl yet!");

                if (useHarmonicEmb)
                    throw std::overflow_error("not impl yet!");

                input_stack_0 = std::make_shared<Conv1D>(
                    this,
                    "input_stack.0",
                    inputChannels,
                    hiddenDims,
                    3,
                    1,
                    1
                );
                input_stack_1 = std::make_shared<GroupNorm1D>(
                    this,
                    "input_stack.1",
                    4,
                    hiddenDims
                );
                input_stack_3 = std::make_shared<Conv1D>(
                    this,
                    "input_stack.3",
                    hiddenDims,
                    hiddenDims,
                    3,
                    1,
                    1
                );

                net = std::make_shared<ConformerNaiveEncoder>(
                    this,
					"net",
                    numLayers,
                    numHeads,
                    hiddenDims,
                    useFaNorm,
                    convOnly
                );

                norm = std::make_shared<LayerNorm1D>(
                    this,
                    "norm",
                    hiddenDims
                );

                output_proj = std::make_shared<Linear>(
                    this,
                    "output_proj",
                    hiddenDims,
                    outputDims
                );
            }

            layerStatus_t Model::Forward(
                CacheTensors& caches
            ) const
            {
                if (auto Ret = input_stack_0->Forward(
                    caches.input,
                    caches.res,
                    caches.col
                )) return Ret;

                if (auto Ret = input_stack_1->Forward(
                    caches.res,
                    caches.mean,
                    caches.var
                )) return Ret;

                if (auto Ret = input_stack_2.Forward(
                    caches.res
                )) return Ret;

                if (auto Ret = input_stack_3->Forward(
                    caches.res,
                    caches.output,
                    caches.col
                )) return Ret;

                if (auto Ret = Transpose::Forward(
                    caches.output,
                    caches.input
                )) return Ret;

                if (auto Ret = net->Forward(
                    caches.input,
                    caches.mean,
                    caches.var,
                    caches.res,
                    caches.output,
                    caches.col
                )) return Ret;

                if (auto Ret = norm->Forward(
                    caches.input,
                    caches.mean,
                    caches.var
                )) return Ret;

                if (auto Ret = output_proj->Forward(
                    caches.input,
                    caches.output
                )) return Ret;

                return SigmoidTensor(caches.output);
            }

		}
	}
}