#include "hip/hip_runtime.h"
#include "upscale.cuh"

static __global__ void upscale_f32(const float * x, float * dst, const int ne00, const int ne00xne01, const int scale_factor) {
    // blockIdx.z: idx of ne02*ne03
    // blockIdx.y: idx of ne01*scale_factor， aka ne1
    // blockIDx.x: idx of ne00*scale_factor / BLOCK_SIZE
    // ne00xne01: ne00 * ne01
    int ne0 = ne00 * scale_factor;
    int nidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (nidx >= ne0) {
        return;
    }
    // operation
    int i00 = nidx / scale_factor;
    int i01 = blockIdx.y / scale_factor;
    int offset_src =
        i00 +
        i01 * ne00 +
        blockIdx.z * ne00xne01;
    int offset_dst =
        nidx +
        blockIdx.y * ne0 +
        blockIdx.z * ne0 * gridDim.y;
    dst[offset_dst] = x[offset_src];
}

static void upscale_f32_cuda(const float * x, float * dst, const int ne00, const int ne01, const int ne02, const int ne03,
                             const int scale_factor, hipStream_t stream) {
    int ne0 = (ne00 * scale_factor);
    int num_blocks = (ne0 + CUDA_UPSCALE_BLOCK_SIZE - 1) / CUDA_UPSCALE_BLOCK_SIZE;
    dim3 gridDim(num_blocks, (ne01 * scale_factor), ne02*ne03);
    upscale_f32<<<gridDim, CUDA_UPSCALE_BLOCK_SIZE, 0, stream>>>(x, dst, ne00, ne00 * ne01, scale_factor);
}

void ggml_cuda_op_upscale(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);
    GGML_ASSERT(src0->ne[3] == 1 && dst->ne[3] == 1); // just 3D tensors

    const int scale_factor = dst->op_params[0];

    upscale_f32_cuda(src0_d, dst_d, src0->ne[0], src0->ne[1], src0->ne[2], src0->ne[3], scale_factor, stream);
}
