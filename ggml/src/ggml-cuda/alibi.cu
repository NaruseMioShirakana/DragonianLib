#include "hip/hip_runtime.h"
#include "alibi.cuh"

static __global__ void alibi_f32(const float * x, float * dst, const int ncols, const int k_rows,
                                 const int n_heads_log2_floor, const float m0, const float m1) {
    const int col = blockDim.x*blockIdx.x + threadIdx.x;

    if (col >= ncols) {
        return;
    }

    const int row = blockDim.y*blockIdx.y + threadIdx.y;
    const int i = row*ncols + col;

    const int k = row/k_rows;

    float m_k;
    if (k < n_heads_log2_floor) {
        m_k = powf(m0, k + 1);
    } else {
        m_k = powf(m1, 2 * (k - n_heads_log2_floor) + 1);
    }

    dst[i] = col * m_k + x[i];
}

static void alibi_f32_cuda(const float * x, float * dst, const int ncols, const int nrows,
                           const int k_rows, const int n_heads_log2_floor, const float m0,
                           const float m1, hipStream_t stream) {
    const dim3 block_dims(CUDA_ALIBI_BLOCK_SIZE, 1, 1);
    const int num_blocks_x = (ncols + CUDA_ALIBI_BLOCK_SIZE - 1) / (CUDA_ALIBI_BLOCK_SIZE);
    const dim3 block_nums(num_blocks_x, nrows, 1);
    alibi_f32<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols, k_rows, n_heads_log2_floor, m0, m1);
}

void ggml_cuda_op_alibi(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t nrows = ggml_nrows(src0);

    //const int n_past = ((int32_t *) dst->op_params)[0];
    const int n_head = ((int32_t *) dst->op_params)[1];
    float max_bias;
    memcpy(&max_bias, (int32_t *) dst->op_params + 2, sizeof(float));

    //GGML_ASSERT(ne01 + n_past == ne00);
    GGML_ASSERT(n_head == ne02);

    const int n_heads_log2_floor = 1 << (int) floor(log2(n_head));

    const float m0 = powf(2.0f, -(max_bias) / n_heads_log2_floor);
    const float m1 = powf(2.0f, -(max_bias / 2.0f) / n_heads_log2_floor);

    alibi_f32_cuda(src0_d, dst_d, ne00, nrows, ne01, n_heads_log2_floor, m0, m1, stream);
}
